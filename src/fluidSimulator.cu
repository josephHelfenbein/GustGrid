#include <iostream>
#include <vector>
#include <unordered_set>
#include <cstring>
#include <hip/hip_runtime.h>

__host__ __device__ static inline int idx3D(int x, int y, int z, int gridSizeX, int gridSizeY){
    return x + y * gridSizeX + z * gridSizeX * gridSizeY;
}

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr<<"CUDA error: "<<hipGetErrorString(err)<<" in "<< __FILE__ <<" on line "<<__LINE__<<std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

constexpr int gridSizeX = 64;
constexpr int gridSizeY = 256;
constexpr int gridSizeZ = 128;
constexpr float worldMinX = -2.0f;
constexpr float worldMaxX = 2.0f;
constexpr float worldMinY = -4.5f;
constexpr float worldMaxY = 4.5f;
constexpr float worldMinZ = -4.0f;
constexpr float worldMaxZ = 4.0f;

constexpr float cellSizeX = (worldMaxX - worldMinX) / gridSizeX;
constexpr float cellSizeY = (worldMaxY - worldMinY) / gridSizeY;
constexpr float cellSizeZ = (worldMaxZ - worldMinZ) / gridSizeZ;

constexpr int maxPressureIterations = 10;
constexpr float pressureTolerance = 1e-4f;

constexpr float thermalDiffusivity = 0.02f;
constexpr float ambientTemperature = 20.0f;
constexpr float coolingRate = 0.001f;
constexpr float heatSourceStrength = 5.0f;

class CudaMemoryPool{
private:
    struct Block{
        void* ptr;
        size_t size;
        bool inUse;
        Block(void* p, size_t s) : ptr(p), size(s), inUse(false) {}
    };
    std::vector<Block> blocks;
    std::unordered_set<void*> allocatedPointers;
public:
    ~CudaMemoryPool(){
        for(auto& block : blocks) if(block.ptr) hipFree(block.ptr);
    }
    void* allocate(size_t size){
        for(auto& block : blocks){
            if(!block.inUse && block.size >= size){
                block.inUse = true;
                allocatedPointers.insert(block.ptr);
                return block.ptr;
            }
        }
        void* newPtr;
        CUDA_CHECK(hipMalloc(&newPtr, size));
        blocks.emplace_back(newPtr, size);
        blocks.back().inUse = true;
        allocatedPointers.insert(newPtr);
        return newPtr;
    }
    void deallocate(void* ptr){
        if(allocatedPointers.find(ptr)==allocatedPointers.end()) return;
        for(auto& block : blocks){
            if(block.ptr == ptr && block.inUse){
                block.inUse = false;
                allocatedPointers.erase(ptr);
                return;
            }
        }
    }
    static CudaMemoryPool& getInstance(){
        static CudaMemoryPool instance;
        return instance;
    }
};

class SimulationMemory{
private:
    float* d_divergence = nullptr;
    float* d_pressure = nullptr;
    float* d_pressureOut = nullptr;
    float* d_residual = nullptr;
    float* d_tempVelocity = nullptr;
    float* d_cgR = nullptr;
    float* d_cgP = nullptr;
    float* d_cgAp = nullptr;
    float* d_cgTemp = nullptr;
    float* d_tempTemperature = nullptr;
    int allocatedGridSize = 0;
public:
    ~SimulationMemory(){
        cleanup();
    }
    void cleanup(){
        if(allocatedGridSize==0) return;
        auto& pool = CudaMemoryPool::getInstance();
        pool.deallocate(d_divergence);
        pool.deallocate(d_pressure);
        pool.deallocate(d_pressureOut);
        pool.deallocate(d_residual);
        pool.deallocate(d_tempVelocity);
        pool.deallocate(d_cgR);
        pool.deallocate(d_cgP);
        pool.deallocate(d_cgAp);
        pool.deallocate(d_cgTemp);
        pool.deallocate(d_tempTemperature);
        allocatedGridSize = 0;
    }
    void ensureAllocated(int numCells){
        if(allocatedGridSize>=numCells) return;
        cleanup();
        auto& pool = CudaMemoryPool::getInstance();
        d_divergence = static_cast<float*>(pool.allocate(numCells * sizeof(float)));
        d_pressure = static_cast<float*>(pool.allocate(numCells * sizeof(float)));
        d_pressureOut = static_cast<float*>(pool.allocate(numCells * sizeof(float)));
        d_residual = static_cast<float*>(pool.allocate(numCells * sizeof(float)));
        d_tempVelocity = static_cast<float*>(pool.allocate(numCells * 3 * sizeof(float)));
        d_cgR = static_cast<float*>(pool.allocate(numCells * sizeof(float)));
        d_cgP = static_cast<float*>(pool.allocate(numCells * sizeof(float)));
        d_cgAp = static_cast<float*>(pool.allocate(numCells * sizeof(float)));
        d_cgTemp = static_cast<float*>(pool.allocate(numCells * sizeof(float)));
        d_tempTemperature = static_cast<float*>(pool.allocate(numCells * sizeof(float)));
        allocatedGridSize = numCells;
    }
    float* getDivergence() { return d_divergence; }
    float* getPressure() { return d_pressure; }
    float* getPressureOut() { return d_pressureOut; }
    float* getResidual() { return d_residual; }
    float* getTempVelocity() { return d_tempVelocity; }
    float* getCGResidual() { return d_cgR; }
    float* getCGSearchDirection() { return d_cgP; }
    float* getCGMatrixVectorProduct() { return d_cgAp; }
    float* getCGTemp() { return d_cgTemp; }
    float* getTempTemperature() { return d_tempTemperature; }
    static SimulationMemory& getInstance() {
        static SimulationMemory instance;
        return instance;
    }
};

__device__ bool isValidFluidCell(int x, int y, int z, int GX, int GY, int GZ, unsigned char* solidGrid) {
    if (x < 0 || x >= GX || y < 0 || y >= GY || z < 0 || z >= GZ) return false;
    return solidGrid[idx3D(x, y, z, GX, GY)] == 0;
}

__global__ void computeDivergenceKernel(
    float* velocity,
    float* divergence,
    unsigned char* solidGrid,
    int GX, int GY, int GZ
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    if (i >= GX || j >= GY || k >= GZ) return;
    int idx = idx3D(i, j, k, GX, GY);
    if(solidGrid[idx] != 0){
        divergence[idx] = 0.0f;
        return;
    }
    float div = 0.0f;
    if(i < GX-1 && i > 0){
        float uRight = isValidFluidCell(i+1, j, k, GX, GY, GZ, solidGrid) ? velocity[idx3D(i+1, j, k, GX, GY) * 3 + 0] : 0.0f;
        float uLeft = isValidFluidCell(i-1, j, k, GX, GY, GZ, solidGrid) ? velocity[idx3D(i-1, j, k, GX, GY) * 3 + 0] : 0.0f;
        div += (uRight - uLeft) / (2.0f * cellSizeX);
    }
    if(j < GY-1 && j > 0){
        float vUp = isValidFluidCell(i, j+1, k, GX, GY, GZ, solidGrid) ? velocity[idx3D(i, j+1, k, GX, GY) * 3 + 1] : 0.0f;
        float vDown = isValidFluidCell(i, j-1, k, GX, GY, GZ, solidGrid) ? velocity[idx3D(i, j-1, k, GX, GY) * 3 + 1] : 0.0f;
        div += (vUp - vDown) / (2.0f * cellSizeY);
    }
    if(k < GZ-1 && k > 0){
        float wFront = isValidFluidCell(i, j, k+1, GX, GY, GZ, solidGrid) ? velocity[idx3D(i, j, k+1, GX, GY) * 3 + 2] : 0.0f;
        float wBack = isValidFluidCell(i, j, k-1, GX, GY, GZ, solidGrid) ? velocity[idx3D(i, j, k-1, GX, GY) * 3 + 2] : 0.0f;
        div += (wFront - wBack) / (2.0f * cellSizeZ);
    }
    divergence[idx] = div;
}

__global__ void pressureJacobianKernel(
    float* pressureIn,
    float* pressureOut,
    float* divergence,
    unsigned char* solidGrid,
    int GX, int GY, int GZ,
    float dt
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    if (i >= GX || j >= GY || k >= GZ) return;
    int idx = idx3D(i, j, k, GX, GY);
    if(solidGrid[idx] != 0){
        pressureOut[idx] = 0.0f;
        return;
    }
    float neighborPressureSum = 0.0f;
    int neighborCount = 0;
    int neighbors[6][3] = {
        {-1, 0, 0},
        {1, 0, 0},
        {0, -1, 0},
        {0, 1, 0},
        {0, 0, -1},
        {0, 0, 1}
    };
    for(int n=0; n<6; n++){
        int ni = i + neighbors[n][0];
        int nj = j + neighbors[n][1];
        int nk = k + neighbors[n][2];
        if(ni >= 0 && ni < GX && nj >= 0 && nj < GY && nk >= 0 && nk < GZ){
            int nidx = idx3D(ni, nj, nk, GX, GY);
            if(solidGrid[nidx] == 0){
                neighborPressureSum += pressureIn[nidx];
                neighborCount++;
            }
        }
    }
    if(neighborCount==0){
        pressureOut[idx] = 0.0f;
        return;
    }
    float beta = 0.6f;
    float avgCellSize = (cellSizeX + cellSizeY + cellSizeZ) / 3.0f;
    float scale = avgCellSize * avgCellSize / dt;
    float newPressure = (neighborPressureSum - divergence[idx] * scale) / neighborCount;
    pressureOut[idx] = (1.0f - beta) * pressureIn[idx] + beta * newPressure;
}

__global__ void subtractPressureGradientKernel(
    float* velocity,
    float* pressure,
    unsigned char* solidGrid,
    int GX, int GY, int GZ,
    float dt
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    if (i >= GX || j >= GY || k >= GZ) return;
    int idx = idx3D(i, j, k, GX, GY);
    if(solidGrid[idx] != 0){
        velocity[idx * 3 + 0] = 0.0f;
        velocity[idx * 3 + 1] = 0.0f;
        velocity[idx * 3 + 2] = 0.0f;
        return;
    }
    float pressureGradientX = 0.0f;
    float pressureGradientY = 0.0f;
    float pressureGradientZ = 0.0f;
    if(i>0 && i<GX-1){
        float pRight = isValidFluidCell(i+1, j, k, GX, GY, GZ, solidGrid) ? pressure[idx3D(i+1, j, k, GX, GY)] : pressure[idx];
        float pLeft = isValidFluidCell(i-1, j, k, GX, GY, GZ, solidGrid) ? pressure[idx3D(i-1, j, k, GX, GY)] : pressure[idx];
        pressureGradientX = (pRight - pLeft) / (2.0f * cellSizeX);
    }
    if(j>0 && j<GY-1){
        float pUp = isValidFluidCell(i, j+1, k, GX, GY, GZ, solidGrid) ? pressure[idx3D(i, j+1, k, GX, GY)] : pressure[idx];
        float pDown = isValidFluidCell(i, j-1, k, GX, GY, GZ, solidGrid) ? pressure[idx3D(i, j-1, k, GX, GY)] : pressure[idx];
        pressureGradientY = (pUp - pDown) / (2.0f * cellSizeY);
    }
    if(k>0 && k<GZ-1){
        float pFront = isValidFluidCell(i, j, k+1, GX, GY, GZ, solidGrid) ? pressure[idx3D(i, j, k+1, GX, GY)] : pressure[idx];
        float pBack = isValidFluidCell(i, j, k-1, GX, GY, GZ, solidGrid) ? pressure[idx3D(i, j, k-1, GX, GY)] : pressure[idx];
        pressureGradientZ = (pFront - pBack) / (2.0f * cellSizeZ);
    }
    velocity[idx * 3 + 0] -= pressureGradientX * dt;
    velocity[idx * 3 + 1] -= pressureGradientY * dt;
    velocity[idx * 3 + 2] -= pressureGradientZ * dt;
}

__global__ void computeResidualKernel(
    float* pressure,
    float* divergence,
    float* residual,
    unsigned char* solidGrid,
    int GX, int GY, int GZ
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    if (i >= GX || j >= GY || k >= GZ) return;
    int idx = idx3D(i, j, k, GX, GY);
    if(solidGrid[idx] != 0){
        residual[idx] = 0.0f;
        return;
    }
    float laplacian = 0.0f;
    float centerPressure = pressure[idx];
    int neighbors[6][3] = {
        {-1, 0, 0},
        {1, 0, 0},
        {0, -1, 0},
        {0, 1, 0},
        {0, 0, -1},
        {0, 0, 1}
    };
    float cellSizes[3] = {cellSizeX, cellSizeY, cellSizeZ};
    for(int n=0; n<6; n++){
        int ni = i + neighbors[n][0];
        int nj = j + neighbors[n][1];
        int nk = k + neighbors[n][2];
        if(ni >= 0 && ni < GX && nj >= 0 && nj < GY && nk >= 0 && nk < GZ){
            int nidx = idx3D(ni, nj, nk, GX, GY);
            if(solidGrid[nidx] == 0){
                int axis = n/2;
                float h = cellSizes[axis];
                laplacian += (pressure[nidx] - centerPressure) / (h * h);
            }
        }
    }
    float residualValue = laplacian - divergence[idx];
    residual[idx] = residualValue * residualValue;
}

__global__ void heatDiffusionKernel(
    float* tempIn,
    float* tempOut,
    float* heatSources,
    unsigned char* solidGrid,
    int GX, int GY, int GZ,
    float dt
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    if (i >= GX || j >= GY || k >= GZ) return;
    int idx = idx3D(i, j, k, GX, GY);
    float temp = tempIn[idx];
    float heatDiffusion = 0.0f;
    int neighbors[6][3] = {
        {-1, 0, 0},
        {1, 0, 0},
        {0, -1, 0},
        {0, 1, 0},
        {0, 0, -1},
        {0, 0, 1}
    };
    float cellSizes[3] = {cellSizeX, cellSizeY, cellSizeZ};
    for(int n=0; n<6; n++){
        int ni = i + neighbors[n][0];
        int nj = j + neighbors[n][1];
        int nk = k + neighbors[n][2];
        if(ni >= 0 && ni < GX && nj >= 0 && nj < GY && nk >= 0 && nk < GZ){
            int nidx = idx3D(ni, nj, nk, GX, GY);
            if(solidGrid[nidx] == 0){
                int axis = n/2;
                float h = cellSizes[axis];
                heatDiffusion += (tempIn[nidx] - temp) / (h * h);
            }
        }
    }
    tempOut[idx] = temp + dt * (
        thermalDiffusivity * heatDiffusion +
        heatSources[idx] * heatSourceStrength -
        coolingRate * (temp - ambientTemperature)
    );
}

__global__ void advectHeatKernel(
    float* tempIn,
    float* tempOut,
    float* velocity,
    unsigned char* solidGrid,
    int GX, int GY, int GZ,
    float dt
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    if (i >= GX || j >= GY || k >= GZ) return;
    int idx = idx3D(i, j, k, GX, GY);
    if(solidGrid[idx] != 0){
        tempOut[idx] = tempIn[idx];
        return;
    }
    float vx = velocity[idx * 3 + 0];
    float vy = velocity[idx * 3 + 1];
    float vz = velocity[idx * 3 + 2];
    float x0 = i - vx * dt / cellSizeX;
    float y0 = j - vy * dt / cellSizeY;
    float z0 = k - vz * dt / cellSizeZ;
    x0 = fmin(fmax(x0, 0.5f), GX - 1.5f);
    y0 = fmin(fmax(y0, 0.5f), GY - 1.5f);
    z0 = fmin(fmax(z0, 0.5f), GZ - 1.5f);
    int xi = int(x0);
    int yi = int(y0);
    int zi = int(z0);
    float fx = x0 - xi;
    float fy = y0 - yi;
    float fz = z0 - zi;
    xi = max(0, min(xi, GX - 2));
    yi = max(0, min(yi, GY - 2));
    zi = max(0, min(zi, GZ - 2));

    float t000 = tempIn[idx3D(xi, yi, zi, GX, GY)];
    float t001 = tempIn[idx3D(xi, yi, zi+1, GX, GY)];
    float t010 = tempIn[idx3D(xi, yi+1, zi, GX, GY)];
    float t011 = tempIn[idx3D(xi, yi+1, zi+1, GX, GY)];
    float t100 = tempIn[idx3D(xi+1, yi, zi, GX, GY)];
    float t101 = tempIn[idx3D(xi+1, yi, zi+1, GX, GY)];
    float t110 = tempIn[idx3D(xi+1, yi+1, zi, GX, GY)];
    float t111 = tempIn[idx3D(xi+1, yi+1, zi+1, GX, GY)];
    float t00 = t000 * (1.0f - fx) + t100 * fx;
    float t01 = t001 * (1.0f - fx) + t101 * fx;
    float t10 = t010 * (1.0f - fx) + t110 * fx;
    float t11 = t011 * (1.0f - fx) + t111 * fx;
    float t0 = t00 * (1.0f - fy) + t10 * fy;
    float t1 = t01 * (1.0f - fy) + t11 * fy;
    tempOut[idx] = t0 * (1.0f - fz) + t1 * fz;
}

__host__ void solvePressureProjection(
    float* d_velocity,
    float* d_pressureField,
    unsigned char* d_solidGrid,
    int GX, int GY, int GZ,
    float dt
){
    const int numCells = GX * GY * GZ;
    auto& simMem = SimulationMemory::getInstance();
    simMem.ensureAllocated(numCells);
    float* d_divergence = simMem.getDivergence();
    float* d_pressure = simMem.getPressure();
    float* d_pressureOut = simMem.getPressureOut();
    float* d_residual = simMem.getResidual();

    CUDA_CHECK(hipMemset(d_pressure, 0, numCells * sizeof(float)));
    CUDA_CHECK(hipMemset(d_pressureOut, 0, numCells * sizeof(float)));

    dim3 block(8, 8, 8);
    dim3 grid(
        (GX + block.x - 1) / block.x,
        (GY + block.y - 1) / block.y,
        (GZ + block.z - 1) / block.z
    );
    computeDivergenceKernel<<<grid, block>>>(
        d_velocity, d_divergence, d_solidGrid, GX, GY, GZ
    );
    CUDA_CHECK(hipDeviceSynchronize());
    float* d_pressure_in = d_pressure;
    float* d_pressure_out = d_pressureOut;
    for(int iter = 0; iter < maxPressureIterations; iter++){
        pressureJacobianKernel<<<grid, block>>>(
            d_pressure_in, d_pressure_out, d_divergence, d_solidGrid, GX, GY, GZ, dt
        );
        CUDA_CHECK(hipDeviceSynchronize());
        if(iter%5==4 || iter == maxPressureIterations-1){
            computeResidualKernel<<<grid, block>>>(
                d_pressure_in, d_divergence, d_residual, d_solidGrid, GX, GY, GZ
            );
            CUDA_CHECK(hipDeviceSynchronize());
            float residualSum = 0.0f;
            float* h_residual = new float[numCells];
            CUDA_CHECK(hipMemcpy(h_residual, d_residual, numCells * sizeof(float), hipMemcpyDeviceToHost));
            for(int i = 0; i < numCells; i++) residualSum += h_residual[i];
            delete[] h_residual;
            float avgResidual = residualSum / numCells;
            if(avgResidual < pressureTolerance){
                std::swap(d_pressure_in, d_pressure_out);
                break;
            }
        }
        std::swap(d_pressure_in, d_pressure_out);
    }
    CUDA_CHECK(hipMemcpy(d_pressureField, d_pressure_in, numCells * sizeof(float), hipMemcpyDeviceToDevice));
    subtractPressureGradientKernel<<<grid, block>>>(
        d_velocity, d_pressure_in, d_solidGrid, GX, GY, GZ, dt
    );
    CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void addFanForcesKernel(
    float* velocity,
    unsigned char* solidGrid,
    float3* fanPos,
    float3* fanDir,
    int numFans,
    float dampeningFactor,
    int GX, int GY, int GZ
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    if (i >= GX || j >= GY || k >= GZ) return;
    int idx = idx3D(i, j, k, GX, GY);
    if(solidGrid[idx] != 0){
        velocity[idx * 3 + 0] = 0.0f;
        velocity[idx * 3 + 1] = 0.0f;
        velocity[idx * 3 + 2] = 0.0f;
        return;
    }
    float worldX = worldMinX + (i + 0.5f) * cellSizeX;
    float worldY = worldMinY + (j + 0.5f) * cellSizeY;
    float worldZ = worldMinZ + (k + 0.5f) * cellSizeZ;
    float3 accum = make_float3(0.0f, 0.0f, 0.0f);
    for(int f = 0; f < numFans; ++f){
        float3 fanPosition = fanPos[f];
        float3 fanDirection = fanDir[f];
        float3 toCell = make_float3(
            worldX - fanPosition.x,
            worldY - fanPosition.y,
            worldZ - fanPosition.z
        );
        float distance = sqrtf(
            toCell.x * toCell.x + 
            toCell.y * toCell.y + 
            toCell.z * toCell.z
        );
        if(distance < 1e-3f) continue;
        float3 toCellNormalized = make_float3(
            toCell.x / distance,
            toCell.y / distance,
            toCell.z / distance
        );
        float alignment = 
            fanDirection.x * toCellNormalized.x +
            fanDirection.y * toCellNormalized.y +
            fanDirection.z * toCellNormalized.z;
        if(alignment > 0.1f){
            float fanRadius = 1.0f;
            float forceMagnitude = 5.0f * alignment / (1.0f + (distance * distance) / (fanRadius * fanRadius));
            accum.x += fanDirection.x * forceMagnitude;
            accum.y += fanDirection.y * forceMagnitude;
            accum.z += fanDirection.z * forceMagnitude;
        }
    }
    velocity[idx * 3 + 0] += accum.x;
    velocity[idx * 3 + 1] += accum.y;
    velocity[idx * 3 + 2] += accum.z;
    const float maxVelocity = 10.0f;
    velocity[idx * 3 + 0] = fminf(fmaxf(velocity[idx * 3 + 0], -maxVelocity), maxVelocity) * dampeningFactor;
    velocity[idx * 3 + 1] = fminf(fmaxf(velocity[idx * 3 + 1], -maxVelocity), maxVelocity) * dampeningFactor;
    velocity[idx * 3 + 2] = fminf(fmaxf(velocity[idx * 3 + 2], -maxVelocity), maxVelocity) * dampeningFactor;
}

__global__ void advectVelocityKernel(
    float* velIn,
    float* velOut,
    unsigned char* solidGrid,
    int GX, int GY, int GZ,
    float dt
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    if (i >= GX || j >= GY || k >= GZ) return;
    int idx = idx3D(i, j, k, GX, GY);
    if(solidGrid[idx] != 0){
        velOut[idx * 3 + 0] = 0.0f;
        velOut[idx * 3 + 1] = 0.0f;
        velOut[idx * 3 + 2] = 0.0f;
        return;
    }
    float vx = velIn[idx * 3 + 0];
    float vy = velIn[idx * 3 + 1];
    float vz = velIn[idx * 3 + 2];
    float advectionStrength = 0.5f;
    float x0 = i - vx * dt * advectionStrength / cellSizeX;
    float y0 = j - vy * dt * advectionStrength / cellSizeY;
    float z0 = k - vz * dt * advectionStrength / cellSizeZ;
    x0 = fminf(fmaxf(x0, 0.5f), GX - 1.5f);
    y0 = fminf(fmaxf(y0, 0.5f), GY - 1.5f);
    z0 = fminf(fmaxf(z0, 0.5f), GZ - 1.5f);
    int xi = int(x0);
    int yi = int(y0);
    int zi = int(z0);
    float fx = x0 - xi;
    float fy = y0 - yi;
    float fz = z0 - zi;
    xi = max(0, min(xi, GX - 2));
    yi = max(0, min(yi, GY - 2));
    zi = max(0, min(zi, GZ - 2));

    for(int comp = 0; comp < 3; comp++){
        float c000 = velIn[idx3D(xi, yi, zi, GX, GY) * 3 + comp];
        float c001 = velIn[idx3D(xi, yi, zi+1, GX, GY) * 3 + comp];
        float c010 = velIn[idx3D(xi, yi+1, zi, GX, GY) * 3 + comp];
        float c011 = velIn[idx3D(xi, yi+1, zi+1, GX, GY) * 3 + comp];
        float c100 = velIn[idx3D(xi+1, yi, zi, GX, GY) * 3 + comp];
        float c101 = velIn[idx3D(xi+1, yi, zi+1, GX, GY) * 3 + comp];
        float c110 = velIn[idx3D(xi+1, yi+1, zi, GX, GY) * 3 + comp];
        float c111 = velIn[idx3D(xi+1, yi+1, zi+1, GX, GY) * 3 + comp];
        
        float c00 = c000 * (1.0f - fx) + c100 * fx;
        float c01 = c001 * (1.0f - fx) + c101 * fx;
        float c10 = c010 * (1.0f - fx) + c110 * fx;
        float c11 = c011 * (1.0f - fx) + c111 * fx;
        
        float c0 = c00 * (1.0f - fy) + c10 * fy;
        float c1 = c01 * (1.0f - fy) + c11 * fy;
        
        velOut[idx * 3 + comp] = c0 * (1.0f - fz) + c1 * fz;
    }
}

extern "C" void runFluidSimulation(
    int gridSizeX, int gridSizeY, int gridSizeZ,
    float* d_velocityField,
    float* d_pressureField,
    unsigned char* d_solidGrid,
    float3* d_fanPositions,
    float3* d_fanDirections,
    float* d_heatSources,
    float* d_temperature,
    int numFans,
    float dt
){
    dim3 block(8, 8, 8);
    dim3 grid(
        (gridSizeX + block.x - 1) / block.x,
        (gridSizeY + block.y - 1) / block.y,
        (gridSizeZ + block.z - 1) / block.z
    );
    const int numCells = gridSizeX * gridSizeY * gridSizeZ;
    auto& simMem = SimulationMemory::getInstance();
    simMem.ensureAllocated(numCells);
    float* d_tempVelocity = simMem.getTempVelocity();
    float* d_tempTemperature = simMem.getTempTemperature();
    addFanForcesKernel<<<grid, block>>>(
        d_velocityField, d_solidGrid, d_fanPositions, d_fanDirections, numFans, 0.95, gridSizeX, gridSizeY, gridSizeZ
    );
    CUDA_CHECK(hipDeviceSynchronize());
    advectVelocityKernel<<<grid, block>>>(
        d_velocityField, d_tempVelocity, d_solidGrid, gridSizeX, gridSizeY, gridSizeZ, dt
    );
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(d_velocityField, d_tempVelocity, numCells * 3 * sizeof(float), hipMemcpyDeviceToDevice));
    solvePressureProjection(
        d_velocityField, d_pressureField, d_solidGrid, gridSizeX, gridSizeY, gridSizeZ, dt
    );
    heatDiffusionKernel<<<grid, block>>>(
        d_temperature, d_tempTemperature, d_heatSources, d_solidGrid, gridSizeX, gridSizeY, gridSizeZ, dt
    );
    CUDA_CHECK(hipDeviceSynchronize());
    advectHeatKernel<<<grid, block>>>(
        d_tempTemperature, d_temperature, d_velocityField, d_solidGrid, gridSizeX, gridSizeY, gridSizeZ, dt
    );
    CUDA_CHECK(hipDeviceSynchronize());
}